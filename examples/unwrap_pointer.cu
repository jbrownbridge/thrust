#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <hip/hip_runtime.h>

int main(void)
{
    size_t N = 10;

    // create a device_ptr 
    thrust::device_ptr<int> dev_ptr = thrust::device_malloc<int>(N);
     
    // extract raw pointer from device_ptr
    int * raw_ptr = thrust::raw_pointer_cast(dev_ptr);

    // use raw_ptr in non-thrust functions
    hipMemset(raw_ptr, 0, N * sizeof(int));

    // free memory
    thrust::device_free(dev_ptr);

    return 0;
}

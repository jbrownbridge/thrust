#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cmath>

// square<T> computes the square of a number f(x) -> x*x
template <typename T>
struct square
{
    __host__ __device__
        T operator()(const T& x) const { 
            return x * x;
        }
};

int main(void)
{
    // initialize host array
    float x[4] = {1.0, 2.0, 3.0, 4.0};

    // transfer to device
    thrust::device_vector<float> d_x(x, x + 4);

    // setup arguments
    square<float>        unary_op;
    thrust::plus<float> binary_op;
    float init = 0;

    // compute norm
    float norm = std::sqrt( thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op) );

    std::cout << norm << std::endl;

    return 0;
}


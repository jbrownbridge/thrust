#include "hip/hip_runtime.h"
#include <thrusttest/unittest.h>
#include <thrust/detail/mpl/math.h>

void TestLog2(void)
{
    unsigned int result;
    
    result = thrust::detail::mpl::math::log2< 1>::value;   ASSERT_EQUAL(result, 0);
    result = thrust::detail::mpl::math::log2< 2>::value;   ASSERT_EQUAL(result, 1);
    result = thrust::detail::mpl::math::log2< 3>::value;   ASSERT_EQUAL(result, 1);
    result = thrust::detail::mpl::math::log2< 4>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2< 5>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2< 6>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2< 7>::value;   ASSERT_EQUAL(result, 2);
    result = thrust::detail::mpl::math::log2< 8>::value;   ASSERT_EQUAL(result, 3);
    result = thrust::detail::mpl::math::log2< 9>::value;   ASSERT_EQUAL(result, 3);
    result = thrust::detail::mpl::math::log2<15>::value;   ASSERT_EQUAL(result, 3);
    result = thrust::detail::mpl::math::log2<16>::value;   ASSERT_EQUAL(result, 4);
    result = thrust::detail::mpl::math::log2<17>::value;   ASSERT_EQUAL(result, 4);
}
DECLARE_UNITTEST(TestLog2);


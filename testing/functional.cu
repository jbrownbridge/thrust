#include <komradetest/unittest.h>
#include <komrade/functional.h>
#include <komrade/transform.h>

#include <functional>
#include <algorithm>
    
const size_t NUM_SAMPLES = 10000;

template <class InputVector, class OutputVector, class Operator, class ReferenceOperator>
void TestUnaryFunctional(void)
{
    typedef typename InputVector::value_type  InputType;
    typedef typename OutputVector::value_type OutputType;
    
    komrade::host_vector<InputType>  std_input = komradetest::random_samples<InputType>(NUM_SAMPLES);
    komrade::host_vector<OutputType> std_output(NUM_SAMPLES);

    InputVector  input = std_input;
    OutputVector output(NUM_SAMPLES);

    komrade::transform(    input.begin(),     input.end(),     output.begin(),          Operator());
    komrade::transform(std_input.begin(), std_input.end(), std_output.begin(), ReferenceOperator());

    ASSERT_EQUAL(output, std_output);
}

template <class InputVector, class OutputVector, class Operator, class ReferenceOperator>
void TestBinaryFunctional(void)
{
    typedef typename InputVector::value_type  InputType;
    typedef typename OutputVector::value_type OutputType;
    
    komrade::host_vector<InputType>  std_input1 = komradetest::random_samples<InputType>(NUM_SAMPLES);
    komrade::host_vector<InputType>  std_input2 = komradetest::random_samples<InputType>(NUM_SAMPLES);
    komrade::host_vector<OutputType> std_output(NUM_SAMPLES);

    // Replace zeros to avoid divide by zero exceptions
    std::replace(std_input2.begin(), std_input2.end(), (InputType) 0, (InputType) 1);

    InputVector input1 = std_input1; 
    InputVector input2 = std_input2; 
    OutputVector output(NUM_SAMPLES);

    komrade::transform(    input1.begin(),     input1.end(),      input2.begin(),     output.begin(),          Operator());
    komrade::transform(std_input1.begin(), std_input1.end(),  std_input2.begin(), std_output.begin(), ReferenceOperator());

    // Note: FP division is not bit-equal, even when nvcc is invoked with --prec-div
    ASSERT_ALMOST_EQUAL(output, std_output);
}



// XXX add bool to list
// Instantiate a macro for all integer-like data types
#define INSTANTIATE_INTEGER_TYPES(Macro, vector_type, operator_name)   \
Macro(vector_type, operator_name, char          )                      \
Macro(vector_type, operator_name, unsigned char )                      \
Macro(vector_type, operator_name, short         )                      \
Macro(vector_type, operator_name, unsigned short)                      \
Macro(vector_type, operator_name, int           )                      \
Macro(vector_type, operator_name, unsigned int  )                      \
Macro(vector_type, operator_name, long          )                      \
Macro(vector_type, operator_name, unsigned long )

// Instantiate a macro for all integer and floating point data types
#define INSTANTIATE_ALL_TYPES(Macro, vector_type, operator_name)       \
INSTANTIATE_INTEGER_TYPES(Macro, vector_type, operator_name)           \
Macro(vector_type, operator_name, float)


// op(T) -> T
#define INSTANTIATE_UNARY_ARITHMETIC_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestUnaryFunctional< komrade::vector_type<data_type>,                                   \
                         komrade::vector_type<data_type>,                                   \
                         komrade::operator_name<data_type>,                                 \
                         std::operator_name<data_type> >();
// XXX revert OutputVector<T> back to bool
// op(T) -> bool
#define INSTANTIATE_UNARY_LOGICAL_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestUnaryFunctional< komrade::vector_type<data_type>,                                \
                         komrade::vector_type<data_type>,                                \
                         komrade::operator_name<data_type>,                              \
                         std::operator_name<data_type> >();
// op(T,T) -> T
#define INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestBinaryFunctional< komrade::vector_type<data_type>,                                   \
                          komrade::vector_type<data_type>,                                   \
                          komrade::operator_name<data_type>,                                 \
                          std::operator_name<data_type> >();
// XXX revert OutputVector<T> back to bool
// op(T,T) -> bool
#define INSTANTIATE_BINARY_LOGICAL_FUNCTIONAL_TEST(vector_type, operator_name, data_type) \
    TestBinaryFunctional< komrade::vector_type<data_type>,                                \
                          komrade::vector_type<data_type>,                                \
                          komrade::operator_name<data_type>,                              \
                          std::operator_name<data_type> >();




// op(T) -> T
#define DECLARE_UNARY_ARITHMETIC_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                          \
void Test##OperatorName##FunctionalHost(void)                                                              \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_ARITHMETIC_FUNCTIONAL_TEST, host_vector,   operator_name);    \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                      \
void Test##OperatorName##FunctionalDevice(void)                                                            \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_ARITHMETIC_FUNCTIONAL_TEST, device_vector, operator_name);    \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T) -> bool
#define DECLARE_UNARY_LOGICAL_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                             \
void Test##OperatorName##FunctionalHost(void)                                                              \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_LOGICAL_FUNCTIONAL_TEST, host_vector,   operator_name);       \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                      \
void Test##OperatorName##FunctionalDevice(void)                                                            \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_UNARY_LOGICAL_FUNCTIONAL_TEST, device_vector, operator_name);       \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T,T) -> T
#define DECLARE_BINARY_ARITHMETIC_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                         \
void Test##OperatorName##FunctionalHost(void)                                                              \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, host_vector,   operator_name);   \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                      \
void Test##OperatorName##FunctionalDevice(void)                                                            \
{                                                                                                          \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, device_vector, operator_name);   \
}                                                                                                          \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T,T) -> T (for integer T only)
#define DECLARE_BINARY_INTEGER_ARITHMETIC_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                     \
void Test##OperatorName##FunctionalHost(void)                                                                  \
{                                                                                                              \
    INSTANTIATE_INTEGER_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, host_vector,   operator_name);   \
}                                                                                                              \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                          \
void Test##OperatorName##FunctionalDevice(void)                                                                \
{                                                                                                              \
    INSTANTIATE_INTEGER_TYPES( INSTANTIATE_BINARY_ARITHMETIC_FUNCTIONAL_TEST, device_vector, operator_name);   \
}                                                                                                              \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);

// op(T,T) -> bool
#define DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(operator_name, OperatorName)                         \
void Test##OperatorName##FunctionalHost(void)                                                           \
{                                                                                                       \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_LOGICAL_FUNCTIONAL_TEST, host_vector,   operator_name);   \
}                                                                                                       \
DECLARE_UNITTEST(Test##OperatorName##FunctionalHost);                                                   \
void Test##OperatorName##FunctionalDevice(void)                                                         \
{                                                                                                       \
    INSTANTIATE_ALL_TYPES( INSTANTIATE_BINARY_LOGICAL_FUNCTIONAL_TEST, device_vector, operator_name);   \
}                                                                                                       \
DECLARE_UNITTEST(Test##OperatorName##FunctionalDevice);




// Create the unit tests
DECLARE_UNARY_ARITHMETIC_FUNCTIONAL_UNITTEST(negate, Negate);
DECLARE_UNARY_LOGICAL_FUNCTIONAL_UNITTEST(logical_not, LogicalNot);

DECLARE_BINARY_ARITHMETIC_FUNCTIONAL_UNITTEST(plus,       Plus      );
DECLARE_BINARY_ARITHMETIC_FUNCTIONAL_UNITTEST(minus,      Minus     );
DECLARE_BINARY_ARITHMETIC_FUNCTIONAL_UNITTEST(multiplies, Multiplies);
DECLARE_BINARY_ARITHMETIC_FUNCTIONAL_UNITTEST(divides,    Divides   );

DECLARE_BINARY_INTEGER_ARITHMETIC_FUNCTIONAL_UNITTEST(modulus, Modulus);

DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(equal_to,      EqualTo     );
DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(not_equal_to,  NotEqualTo  );
DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(greater,       Greater     );
DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(less,          Less        );
DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(greater_equal, GreaterEqual);
DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(less_equal,    LessEqual   );
DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(logical_and,   LogicalAnd  );
DECLARE_BINARY_LOGICAL_FUNCTIONAL_UNITTEST(logical_or,    LogicalOr   );


// Ad-hoc testing for other functionals
template <class Vector>
void TestIdentityFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector input(3);
    input[0] = 0; input[1] = 1; input[2] = 2;

    Vector output(3);

    komrade::transform(input.begin(), input.end(), output.begin(), komrade::identity<T>());

    ASSERT_EQUAL(input, output);
}
DECLARE_VECTOR_UNITTEST(TestIdentityFunctional);



template <class Vector>
void TestMaximumFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector input1(3);
    Vector input2(3);
    input1[0] = 8; input1[1] = 3; input1[2] = 7;
    input2[0] = 5; input2[1] = 6; input2[2] = 9;

    Vector output(3);

    komrade::transform(input1.begin(), input1.end(), 
                       input2.begin(), 
                       output.begin(), 
                       komrade::maximum<T>());

    ASSERT_EQUAL(output[0], 8);
    ASSERT_EQUAL(output[1], 6);
    ASSERT_EQUAL(output[2], 9);
}
DECLARE_VECTOR_UNITTEST(TestMaximumFunctional);

template <class Vector>
void TestMinimumFunctional(void)
{
    typedef typename Vector::value_type T;

    Vector input1(3);
    Vector input2(3);
    input1[0] = 8; input1[1] = 3; input1[2] = 7;
    input2[0] = 5; input2[1] = 6; input2[2] = 9;

    Vector output(3);

    komrade::transform(input1.begin(), input1.end(), 
                       input2.begin(), 
                       output.begin(), 
                       komrade::minimum<T>());

    ASSERT_EQUAL(output[0], 5);
    ASSERT_EQUAL(output[1], 3);
    ASSERT_EQUAL(output[2], 7);
}
DECLARE_VECTOR_UNITTEST(TestMinimumFunctional);


#include <thrusttest/unittest.h>

#if defined(__HIPCC__)

#include <thrust/experimental/arch.h>

using namespace thrust::experimental::arch;

void set_compute_capability(hipDeviceProp_t& properties, int major, int minor)
{
    properties.major = major;
    properties.minor = minor;
}

void set_G80(hipDeviceProp_t& properties)
{
    set_compute_capability(properties, 1, 0);
    properties.multiProcessorCount = 16;
    properties.sharedMemPerBlock   = 16384;
    properties.regsPerBlock        = 8192;
    properties.warpSize            = 32;
    properties.maxThreadsPerBlock  = 512;
}

void set_G84(hipDeviceProp_t& properties)
{
    set_compute_capability(properties, 1, 1);
    properties.multiProcessorCount = 4;
    properties.sharedMemPerBlock   = 16384;
    properties.regsPerBlock        = 8192;
    properties.warpSize            = 32;
    properties.maxThreadsPerBlock  = 512;
}

void set_GT200(hipDeviceProp_t& properties)
{
    set_compute_capability(properties, 1, 3);
    properties.multiProcessorCount = 30;
    properties.sharedMemPerBlock   = 16384;
    properties.regsPerBlock        = 16384;
    properties.warpSize            = 32;
    properties.maxThreadsPerBlock  = 512;
}

void set_func_attributes(hipFuncAttributes& attributes,
                         size_t constSizeBytes,           // Size of constant memory in bytes.
                         size_t localSizeBytes,           // Size of local memory in bytes.
                         int maxThreadsPerBlock,          // Maximum number of threads per block.
                         int numRegs,                     // Number of registers used.
                         size_t sharedSizeBytes)          // Size of shared memory in bytes.
{
    attributes.constSizeBytes     = constSizeBytes;
    attributes.localSizeBytes     = localSizeBytes;
    attributes.maxThreadsPerBlock = maxThreadsPerBlock; 
    attributes.numRegs            = numRegs;
    attributes.sharedSizeBytes    = sharedSizeBytes;
}

void TestMaxActiveThreads(void)
{
    hipDeviceProp_t properties;

    set_compute_capability(properties, 1, 0);
    ASSERT_EQUAL(max_active_threads_per_multiprocessor(properties), 768);
    
    set_compute_capability(properties, 1, 1);
    ASSERT_EQUAL(max_active_threads_per_multiprocessor(properties), 768);
    
    set_compute_capability(properties, 1, 2);
    ASSERT_EQUAL(max_active_threads_per_multiprocessor(properties), 1024);
    
    set_compute_capability(properties, 1, 3);
    ASSERT_EQUAL(max_active_threads_per_multiprocessor(properties), 1024);
}
DECLARE_UNITTEST(TestMaxActiveThreads);


void TestMaxActiveBlocks(void)
{
    hipDeviceProp_t properties;
    hipFuncAttributes attributes;

    // Kernel #1 : Full Occupancy on all devices
    set_func_attributes(attributes, 0, 0, 512, 10, 2048);
    
    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 3);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 3);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 4);
    
    // Kernel #2 : 2/3rds Occupancy on G8x and 100% on GT200
    set_func_attributes(attributes, 0, 0, 512, 16, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 4);
    
    // Kernel #3 : 1/3rds Occupancy on G8x and 75% on GT200
    set_func_attributes(attributes, 0, 0, 512, 20, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 3);
    
    // Kernel #4 : 1/3rds Occupancy on G8x and 50% on GT200
    set_func_attributes(attributes, 0, 0, 512, 21, 2048);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 1);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    
    // Kernel #5 : 2/3rds Occupancy on G8x and 50% on GT200
    set_func_attributes(attributes, 0, 0, 512, 10, 8192);

    set_G80(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_G84(properties);   ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
    set_GT200(properties); ASSERT_EQUAL(max_active_blocks_per_multiprocessor(properties, attributes, 256, 0), 2);
}
DECLARE_UNITTEST(TestMaxActiveBlocks);

#endif // defined(__HIPCC__)

